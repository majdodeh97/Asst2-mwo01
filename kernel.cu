#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

__global__ void mm_kernel(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    // TODO

	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(col >= N || row >= M) return;
	
	float sum = 0f;
	for(unsigned int 1 = 0; i < K; i++){
		sum += A[row*K + i] * B[N*i + col];
	}
	
	C[row*N + col] = sum;
}

void mm_gpu(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO

	float *A_d, *B_d, *C_d;
	
	hipMalloc((void**)&A_d, sizeof(float) * M * K);
	hipMalloc((void**)&B_d, sizeof(float) * K * N);
	hipMalloc((void**)&C_d, sizeof(float) * M * N);




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO

	hipMemcpy(A_d, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
	hipMemcpy(B_d, b, sizeof(double) * K *N, hipMemcpyHostToDevice);




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO

	dim3 numberOfThreadsPerBlock(32, 32);
	dim3 numberOfBlocks((N + numberOfThreadsPerBlock.x - 1) / numberOfThreadsPerBlock.x, (M + numberOfThreadsPerBlock.y - 1) / numberOfThreadsPerBlock.y);

	mm_kernel <<< numberOfBlocks, numberOfThreadsPerBlock >>> (A_d, B_d, C_d, M, N, K);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    // TODO






    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO

	hipFree((void*)A_d);
	hipFree((void*)B_d);
	hipFree((void*)C_d);




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

